#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;
using std::min;

#include <stdlib.h>
#include <math.h>
#include <fstream>
#include <string.h>
#include <time.h>
#include <string.h>
#include <sstream>

#include "hip/hip_runtime.h"

#include "Graph.h"

//#define blockS 16
//#define blockS 32
//define blockS 64
//#define blockS 128
//#define blockS 256
//#define blockS 512
#define blockS 1024

using namespace std;

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
  if (code != hipSuccess){
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

////////////////////////////////////////////////////////////////////////////////

/**
 * Función para actualizar el kernel bidimensional.
 * @param M      Matriz
 * @param nverts numero de vertices
 * @param k      iteracciones
 */
__global__ void floyd1DKernel(int * M, const int nverts, const int k) {
  int ij = threadIdx.x + blockDim.x * blockIdx.x;
  if (ij < nverts * nverts) {
  	int Mij = M[ij];
    int i= ij / nverts;
    int j= ij - i * nverts;
    if (i != j && i != k && j != k) {
  		int Mikj = M[i * nverts + k] + M[k * nverts + j];
    	Mij = (Mij > Mikj) ? Mikj : Mij;
    	M[ij] = Mij;
  	}
  }
}

////////////////////////////////////////////////////////////////////////////////

/**
 * Función para ejecutar el algoritmo de floyd en la GPU usando CUDA.
 * @param h_M             Puntero al inicio de matriz.
 * @param N               Numero de filas
 * @param numBlocks       Numero de bloques que se van a usar
 * @param threadsPorBloque Número de threads por bloque.
 */
void floyd1DGPU(int *h_M, int N, int numBloques, int numThreadsBloque){
    hipError_t err;
    unsigned int sizeMatrix = N * N;
    unsigned int memSize = sizeMatrix * sizeof(int);

    // GPU variables
    int * d_M;

    err = hipMalloc((void **)&d_M, memSize);
  	if (err != hipSuccess) {
  		cout << "ERROR MALLOC d_M" << endl;
  	}

    cout << "CPU: Copiando las matrices de la CPU RAM a la GPU DRAM..." << endl;
    err = hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice);
  	if (err != hipSuccess) {
  		cout << "ERROR COPIA A GPU" << endl;
  	}

    cout << "GPU: Calculando..." << endl;
    for(int k = 0; k < N; k++){
        cout << "KERNEL: " << k << endl;
        floyd1DKernel<<< numBloques, numThreadsBloque >>> (d_M, N, k);
        err = hipGetLastError();
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to launch kernel!\n");
          exit(EXIT_FAILURE);
        }
    }

    cout << "CPU: Copiando los resultados de la GPU DRAM a la CPU RAM..." << endl;

    err = hipMemcpy(h_M, d_M, memSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
  		cout << "ERROR COPIA A GPU" << endl;
  	}

    int i,j;
    for(i=0;i<N;i++) {
      cout << "A["<<i << ",*]= ";
      for(j=0;j<N;j++) {
        if (h_M[i*N+j]==INF)
          cout << "INF";
        else
          cout << h_M[i*N+j];
        if (j < N-1)
          cout << ",";
        else
          cout << endl;
      }
    }

    // Flush all profile data before the application exits

}

void guardarArchivo(std::string outputFile, int n, double t){
  std::ofstream archivo (outputFile.c_str(), std::ios_base::app | std::ios_base::out);
  if (archivo.is_open()){
    std::stringstream ns, ts;
    ns << n;
    ts << t;
    std::string input =  ns.str() + "\t" + ts.str() + "\n";
    archivo << input;
    archivo.close();
  }
  else
    cout << "No se puede abrir el archivo";
}

////////////////////////////////////////////////////////////////////////////////

/**
 * Función para copiar en la matriz h_M el grafo g con tamaño N.
 * @param h_M matriz donde se van a copiar
 * @param g   grafo que se va a copiar
 * @param N   tamaño del grafo.
 */
void copiaGrafo(int * h_M, Graph g, int N){
	for(int i = 0; i<N; i++)
		for(int j = 0; j<N; j++)
			h_M[i * N + j] = g.arista(i,j);
}

void escribeGrafo(int * h_M, Graph g, int N) {
  for(int i = 0; i<N; i++)
    for(int j = 0; j<N; j++)
    g.inserta_arista(i,j, h_M[(i*N)+j]);
}

/**
 * Función principal que ejecuta todo el algoritmo Floyd.
 * @param  argc número de argumentos del programa
 * @param  argv Vector de argumentos del programa
 * @return si ha terminado bien o no la ejecución.
 */
int main(int argc, char **argv){

  if (argc != 2) {
    cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
    return(-1);
  }

  // Get Device Information
  int devID = 0;
  CUDA_CHECK(hipSetDevice(devID));
  CUDA_CHECK(hipGetDevice(&devID));

  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, devID));
  if (deviceProp.computeMode == hipComputeModeProhibited){
      cerr << "Error: La GPU no permite realizar computo ahora mismo, las hebras no pueden usar ::hipSetDevice()." << endl;
      exit(EXIT_SUCCESS);
  }else
    cout << "GPU Device " << devID << ": \"" << deviceProp.name << "\" with compute capability " << deviceProp.major << "." << deviceProp.minor << endl << endl;

  // CPU variables
  Graph G;
  G.lee(argv[1]);
  //G.imprime();

  const unsigned int N = G.vertices;
  const unsigned int sizeMatrix = N * N;
  const unsigned int memSize = sizeMatrix * sizeof(int);
  int * h_M = (int *) malloc(memSize);
  copiaGrafo(h_M, G, N);
  cout << "Grafo copiado en matriz en la RAM" << endl ;

  int numThreadsBloque = blockS;
  int numBloques = (sizeMatrix + numThreadsBloque - 1) / numThreadsBloque;

  cout << "El blockSize es de: " << blockS << endl;
  cout << "El numBloques es de: " << numBloques << endl;
  cout << "El numThreadsBloque es de: " << numThreadsBloque << endl << endl;

  // Calc
  cout << "CPU: Mostrando resultados..." << endl;
  cout << "El Grafo con las distancias de los caminos más cortos es:" << endl << endl;

    /**
    double t1 = clock();
      floyd1DGPU(h_M, N, numBloques, numThreadsBloque);
    double Tgpu = clock();


    **/

    hipError_t err;

    // GPU variables
    int * d_M;

    err = hipMalloc((void **)&d_M, memSize);
    if (err != hipSuccess) {
      cout << "ERROR MALLOC d_M" << endl;
    }

    cout << "CPU: Copiando las matrices de la CPU RAM a la GPU DRAM..." << endl;
    err = hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      cout << "ERROR COPIA A GPU" << endl;
    }

    cout << "GPU: Calculando..." << endl;
    double t1 = clock();
    for(int k = 0; k < N; k++){
        floyd1DKernel<<< numBloques, numThreadsBloque >>> (d_M, N, k);
        err = hipGetLastError();
        if (err != hipSuccess) {
          fprintf(stderr, "Failed to launch kernel!\n");
          exit(EXIT_FAILURE);
        }
    }
    double Tgpu = clock();

    cout << "CPU: Copiando los resultados de la GPU DRAM a la CPU RAM..." << endl;
    err = hipMemcpy(h_M, d_M, memSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      cout << "ERROR COPIA A GPU" << endl;
    }

    int i,j;
    for(i=0;i<N;i++) {
      cout << "A["<<i << ",*]= ";
      for(j=0;j<N;j++) {
        if (h_M[i*N+j]==INF)
          cout << "INF";
        else
          cout << h_M[i*N+j];
        if (j < N-1)
          cout << ",";
        else
          cout << endl;
      }
    }

    Tgpu = (Tgpu-t1)/CLOCKS_PER_SEC;
    cout << "Tiempo gastado GPU = " << Tgpu << endl << endl;

    escribeGrafo(h_M,G,N);

    G.obtenMasLargo();
    G.obtenMasCorto();
    cout << endl;



    // Guardar en el archivo los resultados
    std::string archivo = "output/floyd1D.dat";
    guardarArchivo(archivo, N, Tgpu);

    // Liberando memoria de CPU
    free(h_M);
}
